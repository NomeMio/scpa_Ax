#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include "matriciOpp.h"
#include "../../lib/cuda/cudaHll.cu"




int main(int argc, char *argv[] ) {
   
    struct MatriceHLL *matHll;
    struct MatriceRaw *mat;
    struct Vector *vect;
    struct Vector *result;
    struct Vector *result2;
    struct Vector *resultSerial;
    struct FlatELLMatrix *cudaHllMat;
    hipEvent_t start,stop;

    
  

    if (argc < 3)
    {
        fprintf(stderr, "Usage: %s [matrix-market-filename] [hack]\n", argv[0]);
        exit(1);
    }

    int loadResult = loadMatRaw(argv[1], &mat);
    if (loadResult != 1)
    {
        printf("Errore leggendo la matrice\n");
        return 0;
    }

    int hack = atoi(argv[2]);
    printf("Hack size: %d\n", hack);

    
    
    
    int convResult = convertRawToHll(mat, hack, &matHll);
    if (convResult != 1)
    {
        printf("Error building HLL matrix, error code: %d\n", convResult);
        return convResult;
    }else{
        printf("convertita");
    }
   

   
    int vecResult = generate_random_vector(1, mat->width, &vect);
    if (vecResult != 0)
    {
        printf("Error while creating random vector\n");
        return vecResult;
    }


    int emptyResult = generateEmpty(mat->height, &result);
    if (emptyResult != 0)
    {
        printf("Error while creating result vector\n");
        return emptyResult;
    }

    emptyResult = generateEmpty(mat->height, &resultSerial);
    if (emptyResult != 0)
    {
        printf("Error while creating result vectorSerial\n");
        return emptyResult;
    }

    printf("\n dimensione resultSeires: %d",resultSerial->righe);

    

    emptyResult = generateEmpty(mat->height, &result2);
    if (emptyResult != 0)
    {
        printf("Error while creating result vectorSerial\n");
        return emptyResult;
    }

 

    int flatHll = convertHLLToFlatELL(&matHll, &cudaHllMat);
    if (emptyResult != 0)
    {
        printf("Error while converting to flat format result vector\n");
        return emptyResult;
    }

  
    //printFlatELLMatrix(&cudaHllMat);

    int total_rows = 0;
    for (int i = 0; i < cudaHllMat->numBlocks; i++) {
        total_rows += cudaHllMat->block_rows[i];
    }


    
    double *d_values_flat;
    hipMalloc((void**)&d_values_flat, sizeof(double) * cudaHllMat->total_values);
    hipMemcpy(d_values_flat, cudaHllMat->values_flat, sizeof(double) * cudaHllMat->total_values, hipMemcpyHostToDevice);

    
    // Array degli indici di colonna flattenati
    int *d_col_indices_flat;
    hipMalloc((void**)&d_col_indices_flat, sizeof(int) * cudaHllMat->total_values);
    hipMemcpy(d_col_indices_flat, cudaHllMat->col_indices_flat, sizeof(int) * cudaHllMat->total_values, hipMemcpyHostToDevice);

    

    // Array degli offset di inizio di ogni blocco
    int *d_block_offsets;
    hipMalloc((void**)&d_block_offsets, sizeof(int) * cudaHllMat->numBlocks);
    hipMemcpy(d_block_offsets, cudaHllMat->block_offsets, sizeof(int) * cudaHllMat->numBlocks, hipMemcpyHostToDevice);

   

    // Array del numero massimo di non zero per riga per ogni blocco (MAXNZ)
    int *d_block_nnz;
    hipMalloc((void**)&d_block_nnz, sizeof(int) * cudaHllMat->numBlocks);
    hipMemcpy(d_block_nnz, cudaHllMat->block_nnz, sizeof(int) * cudaHllMat->numBlocks, hipMemcpyHostToDevice);

    

    // Array del numero di righe effettive per ogni blocco
    int *d_block_rows;
    hipMalloc((void**)&d_block_rows, sizeof(int) * cudaHllMat->numBlocks);
    hipMemcpy(d_block_rows, cudaHllMat->block_rows, sizeof(int) * cudaHllMat->numBlocks, hipMemcpyHostToDevice);

 

    
    // 2. Aggiornamento della struttura host con i puntatori device
    cudaHllMat->values_flat    = d_values_flat;
    cudaHllMat->col_indices_flat = d_col_indices_flat;
    cudaHllMat->block_offsets  = d_block_offsets;
    cudaHllMat->block_nnz      = d_block_nnz;
    cudaHllMat->block_rows     = d_block_rows;

    // 3. Allocazione della struttura sulla GPU e copia della struttura aggiornata
    struct FlatELLMatrix *d_mat;
    hipMalloc((void**)&d_mat, sizeof(struct FlatELLMatrix));
    hipMemcpy(d_mat, cudaHllMat, sizeof(struct FlatELLMatrix), hipMemcpyHostToDevice);

    
  
    double *d_vettore;
    int righex=vect->righe;
    hipMalloc((void**)&d_vettore, sizeof(double) * vect->righe);
    hipMemcpy(d_vettore, vect->vettore, sizeof(double) * vect->righe, hipMemcpyHostToDevice);

    // 2. Aggiorna il campo 'vettore' della struttura host per puntare all'array allocato sulla GPU
    double *temp=vect->vettore;
    vect->vettore = d_vettore;

    // 3. Alloca la struttura 'Vector' su GPU e copia la struttura aggiornata
    struct Vector *d_vect;
    hipMalloc((void**)&d_vect, sizeof(struct Vector));
    hipMemcpy(d_vect, vect, sizeof(struct Vector), hipMemcpyHostToDevice);


    double *d_result_vettore;
    hipMalloc((void**)&d_result_vettore, sizeof(double) * result->righe);
 

    // 2. Aggiorna il campo 'vettore' della struttura host per 'result'
    result->vettore = d_result_vettore;

    // 3. Alloca la struttura 'Vector' su GPU e copia la struttura aggiornata
    struct Vector *d_result;
    hipMalloc((void**)&d_result, sizeof(struct Vector));
    hipMemcpy(d_result, result, sizeof(struct Vector), hipMemcpyHostToDevice);

   
   int *d_numBlocks;
    hipMalloc(&d_numBlocks, sizeof(int));
    hipMemcpy(d_numBlocks, &cudaHllMat->numBlocks, sizeof(int), hipMemcpyHostToDevice);
   
    
   
   

    int block_size = 32;
    int num_threads = matHll->numBlocks * hack;
    int grid_size = (num_threads + block_size - 1) / block_size;

    printf("\nrighe: %d", righex);


    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);


    matvec_flatell_kernel2_reduction<<<grid_size, block_size,num_threads * sizeof(double)>>>(d_mat,d_vettore,d_result_vettore,hack);

    //matvec_flatell_kernel4<<<grid_size, block_size, sharedMemSize>>>(d_mat,d_vettore,d_result_vettore,hack,righex);

   /*
   matvec_flatell_kernel3_safe<<<grid_size, block_size>>>(
    d_values_flat,
    d_col_indices_flat,
    d_block_offsets,
    d_block_nnz,  
    d_block_rows,
    d_vettore,
    d_result_vettore,
    d_numBlocks,  
    hack,
    vect->righe);
   
   */

    hipEventRecord(stop, 0);

    hipEventSynchronize(stop);

    float time_ms;
    hipEventElapsedTime(&time_ms, start, stop);

    
    double time_sec = time_ms / 1000.0;

    double totalFLOPs = 2.0 * cudaHllMat->total_values;

    double gflops = totalFLOPs / (time_sec * 1e9);

    printf("Tempo medio del kernel: %f s\n", time_sec);
    printf("GFLOPS: %lf\n", gflops);


    hipError_t memcopy;
     // Copia del risultato dalla GPU alla CPU
    memcopy=hipMemcpy(result2->vettore, d_result_vettore, result2->righe * sizeof(double), hipMemcpyDeviceToHost);
    if (memcopy!=hipSuccess) {
        printf("errore");
    }   


    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(d_values_flat);
    hipFree(d_col_indices_flat);
    hipFree(d_vect);
    hipFree(d_result);
    hipFree(d_block_offsets);
    hipFree(d_block_nnz);
    hipFree(d_block_rows);

   
  
   
    vect->vettore=temp;
    
  
    
    double time2=0;

    int multResult = hllMultWithTime(&serialMultiplyHLL,matHll, vect, resultSerial, &time2);
    if (multResult != 0)
    {
        printf("Error in serialMultiply, error code: %d\n", multResult);
        return multResult;
    }
    


   for(int i=0;i < (result2->righe) ;i++){

        if(result2->vettore[i]!=resultSerial->vettore[i]){
            printf("\n: valori diversi (%lf vs %lf)\n", result2->vettore[i], resultSerial->vettore[i]);
        }

   }
    
    int check=areVectorsEqual(result2,resultSerial);
    if(check!=0){
        printf("the vector are different");
    }
    

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Errore nel lancio del kernel: %s\n", hipGetErrorString(err));
        return -1;
    }

    
     
    

    return 0;
}