#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloKernel() {
    printf("Ciao da CUDA thread %d\n", threadIdx.x);
}

int main() {
    printf("Starting kernel...\n");

    // Imposta una dimensione maggiore per il buffer di printf (opzionale)
    hipDeviceSetLimit(hipLimitPrintfFifoSize, 1048576);

    // Lancia il kernel con 1 blocco di 10 thread
    helloKernel<<<1, 10>>>();

    // Controlla subito errori di lancio
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Kernel launch error: %s\n", hipGetErrorString(err));
    }

    // Sincronizza e controlla errori
    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        printf("CUDA error after synchronization: %s\n", hipGetErrorString(err));
    }

    // Forza lo svuotamento degli output
    fflush(stdout);

    printf("Kernel execution finished.\n");
    return 0;
}