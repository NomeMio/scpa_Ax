#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include "matriciOpp.h"
#include "cuda_alex.h"
#include "cuda_luca.h"
#define WARP_SIZE 32

#define DEBUG 0
  
void freeFlatHLL(FlatELLMatrix **flatMat){
    struct FlatELLMatrix *mat=*flatMat;
    free(mat->values_flat);
    free(mat->col_indices_flat);
    free(mat->block_offsets);
    free(mat->block_nnz);
    free(mat->block_rows);

}
int convertHLLToFlatELL(MatriceHLL **H, FlatELLMatrix **flatMat)
{
    // Allocazione della struttura FlatELLMatrix
    *flatMat = (FlatELLMatrix *)malloc(sizeof(FlatELLMatrix));
    if (!(*flatMat))
    {
        perror("Errore di allocazione della struttura FlatELLMatrix");
        return -1;
    }

    int numBlocks = (*H)->numBlocks;
    (*flatMat)->numBlocks = numBlocks;

    // Calcola il numero totale di elementi da allocare
    int total = 0;
    for (int b = 0; b < numBlocks; b++)
    {
        ELLPACK_Block *block = (*H)->blocks[b];
        if (block)
        {
            total += block->M * block->MAXNZ;
        }
    }
    (*flatMat)->total_values = total;

    // Allocazione degli array per il formato Flat ELLPACK
    (*flatMat)->values_flat = (double *)malloc(total * sizeof(double));
    (*flatMat)->col_indices_flat = (int *)malloc(total * sizeof(int));
    (*flatMat)->block_offsets = (int *)malloc(numBlocks * sizeof(int));
    (*flatMat)->block_nnz = (int *)malloc(numBlocks * sizeof(int));
    (*flatMat)->block_rows = (int *)malloc(numBlocks * sizeof(int));
    (*flatMat)->hack = (*H)->HackSize;


    if (!(*flatMat)->values_flat || !(*flatMat)->col_indices_flat ||
        !(*flatMat)->block_offsets || !(*flatMat)->block_nnz || !(*flatMat)->block_rows)
    {
        perror("Errore di allocazione negli array Flat ELLPACK");
        return -1;
    }

    int offset = 0;
    for (int b = 0; b < numBlocks; b++)
    {
        ELLPACK_Block *block = (*H)->blocks[b];
        if (!block)
            continue;

        int M = block->M;
        int MAXNZ = block->MAXNZ;

        // Salva i metadati per il blocco corrente
        (*flatMat)->block_offsets[b] = offset;
        (*flatMat)->block_nnz[b] = MAXNZ;
        (*flatMat)->block_rows[b] = M;

        // Copia dei dati: si copia in ordine colonna-per-riga.
        // L'elemento nella riga i e nella "colonna slot" j del blocco
        // viene memorizzato a: offset + j * M + i.
        for (int i = 0; i < M; i++)
        {
            for (int j = 0; j < MAXNZ; j++)
            {
                int dst_idx = offset + j * M + i;
                int src_idx = i * MAXNZ + j; // Gli array JA e AS sono in ordine riga-per-riga
                (*flatMat)->values_flat[dst_idx] = block->AS[src_idx];
                (*flatMat)->col_indices_flat[dst_idx] = block->JA[src_idx];
            }
        }
        offset += M * MAXNZ;
    }
    printf("finished parsing hll flat matrix\n");

    return 0;
}




void printFlatELLMatrix(FlatELLMatrix **flatMat)
{
    if (flatMat == NULL || *flatMat == NULL)
    {
        printf("La struttura FlatELLMatrix è NULL.\n");
        return;
    }

    FlatELLMatrix *F = *flatMat;
    printf("Flat ELLPACK Matrix:\n");
    printf("Total values: %d, numBlocks: %d\n", F->total_values, F->numBlocks);

    // Scorre ciascun blocco
    for (int b = 0; b < F->numBlocks; b++)
    {
        int offset = F->block_offsets[b];
        int rows = F->block_rows[b];
        int maxnz = F->block_nnz[b];

        printf("Block %d: offset = %d, rows = %d, MAXNZ = %d\n", b, offset, rows, maxnz);

        // Per ogni riga del blocco
        for (int i = 0; i < rows; i++)
        {
            // Per ogni "slot" nella riga (fino a MAXNZ)
            for (int j = 0; j < maxnz; j++)
            {

                int idx = offset + j * rows + i;
                printf("[col=%d, val=%f] ", F->col_indices_flat[idx], F->values_flat[idx]);
            }
            printf("\n");
        }
        printf("\n");
    }
}
__global__ void matvec_flatell_kernel(struct FlatELLMatrix *dMat, double *x, double *y, int hack_size) {
    int global_row = blockIdx.x * blockDim.x + threadIdx.x; 

    
    if (global_row >= dMat->numBlocks * hack_size) return;

    // Trova a quale blocco appartiene questa riga
    int block_id = global_row / hack_size;
    if (block_id >= dMat->numBlocks) return;

    int block_start = dMat->block_offsets[block_id];   // Offset del blocco
    int rows_in_block = dMat->block_rows[block_id];    // Righe nel blocco

    // Riga locale nel blocco
    int local_row = global_row % hack_size;
    if (local_row >= rows_in_block) return;

    double sum = 0.0;
    int max_nnz = dMat->block_nnz[block_id];  // NNZ massimo per riga nel blocco

    // Moltiplicazione matrice-vettore per la riga corrente
    for (int j = 0; j < max_nnz; j++) {
        int col = dMat->col_indices_flat[block_start + j * rows_in_block + local_row];
        if (col >= 0) {
            sum += dMat->values_flat[block_start + j * rows_in_block + local_row] * x[col];
        }
    }

    y[global_row] = sum;
}


__global__ void matvec_flatell_kernel_2(FlatELLMatrix *dMat, double *x, double *y, int hack_size, int N) {
    extern __shared__ double shared_x[];
    int tid = threadIdx.x;
    int global_row = blockIdx.x * blockDim.x + tid;
    int block_size = blockDim.x;

    if (global_row >= dMat->numBlocks * hack_size) return;

    // Trova a quale blocco appartiene questa riga
    int block_id = global_row / hack_size;
    if (block_id >= dMat->numBlocks) return;

    int block_start = dMat->block_offsets[block_id];   // Offset del blocco
    int rows_in_block = dMat->block_rows[block_id];    // Righe nel blocco

    // Riga locale nel blocco
    int local_row = global_row % hack_size;
    if (local_row >= rows_in_block) return;

    // Caricamento di una porzione di x in memoria condivisa
    if (tid < block_size && tid < N) {
        shared_x[tid] = x[tid];
    }
    __syncthreads();

    double sum = 0.0;
    int max_nnz = dMat->block_nnz[block_id];  // NNZ massimo per riga nel blocco

   
    for (int j = 0; j < max_nnz; j++) {
        int col = dMat->col_indices_flat[block_start + j * rows_in_block + local_row];
        if (col >= 0) {
            // Accesso a x dalla shared memory se l'indice è nel range caricato
            double x_val = (col < block_size && col < N) ? shared_x[col] : x[col];
            sum += dMat->values_flat[block_start + j * rows_in_block + local_row] * x_val;
        }
    }

    y[global_row] = sum;
}




__global__ void matvec_flatell_kernel_v3(FlatELLMatrix *dMat, double *x, double *y, int hack_size,int total_row) {

    int thread_id = blockIdx.x * blockDim.x + threadIdx.x;  // ID del thread
    int warp_id = thread_id >> 5;  // Ogni warp lavora su una riga (thread_id / 32)
    int lane = thread_id & 31;     // ID del thread dentro la warp (0-31)



    if (warp_id >= total_row) return;

    // Calcolare a quale hack appartiene questa riga (ogni hack corrisponde a un blocco)
    int block_id = warp_id / hack_size;
    int local_row = warp_id % hack_size;
    int rows_in_block = dMat->block_rows[block_id];

    if (local_row >= rows_in_block) return;  // Assicurarsi che non si esca dai limiti della riga

    int block_start = dMat->block_offsets[block_id];  // Offset del blocco
    int max_nnz_per_row = dMat->block_nnz[block_id]; // Max NNZ per riga nel blocco
    double sum = 0.0;

    for (int j = lane; j < max_nnz_per_row; j += 32) {
        
        int flat_idx = block_start + j * rows_in_block + local_row;

        int col = dMat->col_indices_flat[flat_idx];

        // Controlla se è un padding (spesso indicato con col < 0)
        if (col >= 0) {
            double val = dMat->values_flat[flat_idx];
            sum += val * x[col]; // Accumula il prodotto
        }
    }

    int width=32;
    // Riduzione a livello di warp per sommare i risultati parziali
    for (int offset = width >> 1; offset > 0; offset >>= 1) {
        sum += __shfl_down_sync(0xFFFFFFFF, sum, offset,width);
    }

    // Il primo thread della warp scrive il risultato finale
    if (lane == 0) {
        y[warp_id] = sum;
    }
}
__global__ void matvec_flatell_kernel_warpCOlonne(FlatELLMatrix *dMat, double *x, double *y, int hack_size,int total_blocks) {

    int thread_id = blockIdx.x * blockDim.x + threadIdx.x;  // ID del thread
    int warp_id = thread_id >> 5; 
    int lane = thread_id & 31;    
    if (warp_id >= total_blocks ) return; 
    int row=dMat->block_rows[warp_id];
    if(row<=lane)return;
    int block_start = dMat->block_offsets[warp_id];  // Offset del blocco
    int max_nnz_per_row = dMat->block_nnz[warp_id]; // Max NNZ per riga nel blocco
    double sum = 0.0;
    for (int j = 0; j < max_nnz_per_row-1  ; j += 1) {
        int flat_idx = block_start + j * row + lane;

        int col = dMat->col_indices_flat[flat_idx]; //evitabile?
        double molt=x[col];
        double val = dMat->values_flat[flat_idx];
        sum += val *  molt;// Accumula il prodotto
    }
    int flat_idx = block_start + (max_nnz_per_row-1) * row + lane;
    
    int col = dMat->col_indices_flat[flat_idx]; //evitabile?
    double molt=x[col];
    double val = dMat->values_flat[flat_idx];
    sum += val *  molt;// Accumula il prodotto
    y[hack_size*warp_id+lane]=sum;
}
int * allocVectorGpuInt(Vector *vect){
    int *d_vettore;
    int righex=vect->righe;
    hipMalloc((void**)&d_vettore, sizeof(int) * vect->righe);
    hipMemcpy(d_vettore, vect->vettore, sizeof(int) * vect->righe, hipMemcpyHostToDevice);
    return d_vettore;    
}

typedef struct FlatEllAllocated{
    struct FlatELLMatrix * gpu;
    struct FlatELLMatrix * pointersToGpu;
}FlatEllAllocated;

// Funzione per controllare gli errori CUDA e terminare in caso di fallimento
static void HandleCudaError(hipError_t err, const char *file, int line) {
    if (err != hipSuccess) {
        fprintf(stderr, "%s in %s at line %d\n", hipGetErrorString(err), file, line);
        exit(EXIT_FAILURE);
    }
}
#define HANDLE_CUDA_ERROR(err) (HandleCudaError(err, __FILE__, __LINE__))

double * allocVectorGpuDouble(Vector *vect){
    double *d_vettore;
    hipError_t err;

    err = hipMalloc((void**)&d_vettore, sizeof(double) * vect->righe);
    HANDLE_CUDA_ERROR(err);

    err = hipMemcpy(d_vettore, vect->vettore, sizeof(double) * vect->righe, hipMemcpyHostToDevice);
    HANDLE_CUDA_ERROR(err);

    return d_vettore;
}

struct FlatEllAllocated allocateFlatHll(struct FlatELLMatrix *cudaHllMat){
    hipError_t err;
    double *d_values_flat;
    err = hipMalloc((void**)&d_values_flat, sizeof(double) * cudaHllMat->total_values);
    HANDLE_CUDA_ERROR(err);
    err = hipMemcpy(d_values_flat, cudaHllMat->values_flat, sizeof(double) * cudaHllMat->total_values, hipMemcpyHostToDevice);
    HANDLE_CUDA_ERROR(err);

    int *d_col_indices_flat;
    err = hipMalloc((void**)&d_col_indices_flat, sizeof(int) * cudaHllMat->total_values);
    HANDLE_CUDA_ERROR(err);
    err = hipMemcpy(d_col_indices_flat, cudaHllMat->col_indices_flat, sizeof(int) * cudaHllMat->total_values, hipMemcpyHostToDevice);
    HANDLE_CUDA_ERROR(err);

    int *d_block_offsets;
    err = hipMalloc((void**)&d_block_offsets, sizeof(int) * cudaHllMat->numBlocks);
    HANDLE_CUDA_ERROR(err);
    err = hipMemcpy(d_block_offsets, cudaHllMat->block_offsets, sizeof(int) * cudaHllMat->numBlocks, hipMemcpyHostToDevice);
    HANDLE_CUDA_ERROR(err);

    int *d_block_nnz;
    err = hipMalloc((void**)&d_block_nnz, sizeof(int) * cudaHllMat->numBlocks);
    HANDLE_CUDA_ERROR(err);
    err = hipMemcpy(d_block_nnz, cudaHllMat->block_nnz, sizeof(int) * cudaHllMat->numBlocks, hipMemcpyHostToDevice);
    HANDLE_CUDA_ERROR(err);

    int *d_block_rows;
    err = hipMalloc((void**)&d_block_rows, sizeof(int) * cudaHllMat->numBlocks);
    HANDLE_CUDA_ERROR(err);
    err = hipMemcpy(d_block_rows, cudaHllMat->block_rows, sizeof(int) * cudaHllMat->numBlocks, hipMemcpyHostToDevice);
    HANDLE_CUDA_ERROR(err);

    struct FlatELLMatrix *cudaHllMatG;
    cudaHllMatG = (struct FlatELLMatrix *)malloc(sizeof(struct FlatELLMatrix));
    if (cudaHllMatG == NULL) {
        fprintf(stderr, "Errore di allocazione memoria host per cudaHllMatG in %s at line %d\n", __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }
    cudaHllMatG->values_flat    = d_values_flat;
    cudaHllMatG->col_indices_flat = d_col_indices_flat;
    cudaHllMatG->block_offsets  = d_block_offsets;
    cudaHllMatG->block_nnz      = d_block_nnz;
    cudaHllMatG->block_rows     = d_block_rows;
    cudaHllMatG->hack           = cudaHllMat->hack;
    cudaHllMatG->total_values   = cudaHllMat->total_values;
    cudaHllMatG->numBlocks      = cudaHllMat->numBlocks;

    struct FlatELLMatrix *d_mat;
    err = hipMalloc((void**)&d_mat, sizeof(struct FlatELLMatrix));
    HANDLE_CUDA_ERROR(err);
    err = hipMemcpy(d_mat, cudaHllMatG, sizeof(struct FlatELLMatrix), hipMemcpyHostToDevice);
    HANDLE_CUDA_ERROR(err);

    return (struct FlatEllAllocated) {.gpu=d_mat,.pointersToGpu=cudaHllMatG};
}


int invokeKernel1(struct Vector *vect,
        struct Vector *result,
        struct FlatELLMatrix *cudaHllMat, struct MatriceHLL *matHll,int hack, double *time ,int blockS){ //TODO:da togliere matrice hll

        hipEvent_t start,stop;
        
    // alloco le strutture cuda
        struct FlatEllAllocated d_mat=allocateFlatHll(cudaHllMat);
        double *d_result_vettore=allocVectorGpuDouble(result);
        double *d_vettore=allocVectorGpuDouble(vect);
                
    // calcolo i blocchi necessari per l'inferenza
        int block_size = blockS;
        int num_threads = matHll->numBlocks * hack;
        int grid_size = (num_threads + block_size - 1) / block_size;
    
        // startp il timer
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start, 0);

        matvec_flatell_kernel<<<grid_size, block_size>>>(d_mat.gpu,d_vettore,d_result_vettore,hack);

        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            fprintf(stderr, "Errore nel lancio del kernel: %s\n", hipGetErrorString(err));
            return -1;
        }
        // calcolo il tempo del esecuznione
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        float time_ms;
        hipEventElapsedTime(&time_ms, start, stop);
        double time_sec = time_ms / 1000.0;
        *time=time_sec;


        hipError_t memcopy;
        memcopy=hipMemcpy(result->vettore, d_result_vettore, result->righe * sizeof(double), hipMemcpyDeviceToHost);
        if (memcopy!=hipSuccess) {
            printf("errore");
        }   

        hipEventDestroy(start);
        hipEventDestroy(stop);
        hipFree(d_mat.pointersToGpu->values_flat);
        hipFree(d_mat.pointersToGpu->col_indices_flat );
        hipFree(d_result_vettore);
        hipFree(d_mat.pointersToGpu->block_offsets);
        hipFree(d_mat.pointersToGpu->block_nnz);
        hipFree(d_mat.pointersToGpu->block_rows);
        free(d_mat.pointersToGpu);

        *time=time_sec;


        
        return 0;
    
    }

int invokeKernel2(struct Vector *vect,
    struct Vector *result,
    struct FlatELLMatrix *cudaHllMat, struct MatriceHLL *matHll,int hack,double* time,int blockS){

        hipEvent_t start,stop;
        
    // alloco le strutture cuda
        struct FlatEllAllocated d_mat=allocateFlatHll(cudaHllMat);
        double *d_result_vettore=allocVectorGpuDouble(result);
        double *d_vettore=allocVectorGpuDouble(vect);
                
   
    int block_size = blockS;
    int num_threads = matHll->numBlocks * hack;
    int grid_size = (num_threads + block_size - 1) / block_size;
    size_t shared_mem_size = num_threads * sizeof(double);

   
   
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);


    matvec_flatell_kernel_2<<<grid_size, block_size,1024>>>(d_mat.gpu,d_vettore,d_result_vettore,hack,vect->righe);


   hipError_t err = hipGetLastError();
        HANDLE_CUDA_ERROR(err);
        // calcolo il tempo del esecuznione
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        float time_ms;
        hipEventElapsedTime(&time_ms, start, stop);
        double time_sec = time_ms / 1000.0;
        *time=time_sec;


        hipError_t memcopy;
        memcopy=hipMemcpy(result->vettore, d_result_vettore, result->righe * sizeof(double), hipMemcpyDeviceToHost);
        HANDLE_CUDA_ERROR(memcopy);

        hipEventDestroy(start);
        hipEventDestroy(stop);
        hipFree(d_mat.pointersToGpu->values_flat);
        hipFree(d_mat.pointersToGpu->col_indices_flat );
        hipFree(d_result_vettore);
        hipFree(d_mat.pointersToGpu->block_offsets);
        hipFree(d_mat.pointersToGpu->block_nnz);
        hipFree(d_mat.pointersToGpu->block_rows);
        free(d_mat.pointersToGpu);

        *time=time_sec;


        
        return 0;
  
}


int invokeKernel3(struct Vector *vect,
    struct Vector *result,
    struct FlatELLMatrix *cudaHllMat, struct MatriceHLL *matHll,int hack,double* time,int blockS ){

        hipEvent_t start,stop;
        
    // alloco le strutture cuda
        struct FlatEllAllocated d_mat=allocateFlatHll(cudaHllMat);
        double *d_result_vettore=allocVectorGpuDouble(result);
        double *d_vettore=allocVectorGpuDouble(vect);
                
   
    int block_size = blockS;
    int num_threads = matHll->numBlocks * hack;
    int grid_size = (num_threads + block_size - 1) / block_size;
    size_t shared_mem_size = num_threads * sizeof(double);

   
   
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    int threadsPerBlock = blockS;
    int numBlocks = matHll->totalRows;
   

    matvec_flatell_kernel_v3<<<numBlocks, threadsPerBlock>>>(d_mat.gpu,d_vettore,d_result_vettore,hack,matHll->totalRows);


     hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            fprintf(stderr, "Errore nel lancio del kernel: %s\n", hipGetErrorString(err));
            return -1;
        }
        // calcolo il tempo del esecuznione
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        float time_ms;
        hipEventElapsedTime(&time_ms, start, stop);
        double time_sec = time_ms / 1000.0;
        *time=time_sec;


        hipError_t memcopy;
        memcopy=hipMemcpy(result->vettore, d_result_vettore, result->righe * sizeof(double), hipMemcpyDeviceToHost);
        if (memcopy!=hipSuccess) {
            printf("errore");
        }   

        hipEventDestroy(start);
        hipEventDestroy(stop);
        hipFree(d_mat.pointersToGpu->values_flat);
        hipFree(d_mat.pointersToGpu->col_indices_flat );
        hipFree(d_result_vettore);
        hipFree(d_mat.pointersToGpu->block_offsets);
        hipFree(d_mat.pointersToGpu->block_nnz);
        hipFree(d_mat.pointersToGpu->block_rows);
        free(d_mat.pointersToGpu);

        *time=time_sec;


        
        return 0;

  
}

int invokeKernelWarpColonne(struct Vector *vect,
    struct Vector *result,
    struct FlatELLMatrix *cudaHllMat, struct MatriceHLL *matHll,int hack,double* time,int blockS ){
        hipEvent_t start,stop;
        
        struct FlatEllAllocated d_mat=allocateFlatHll(cudaHllMat);
        double *d_result_vettore=allocVectorGpuDouble(result);
        double *d_vettore=allocVectorGpuDouble(vect);
                
   
    int block_size = blockS;
    int num_threads = matHll->numBlocks*32;
    int grid_size = (num_threads + block_size - 1) / block_size;

   
   
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    int threadsPerBlock = blockS;
    //int numBlocks = matHll->totalRows;
    matvec_flatell_kernel_warpCOlonne<<<grid_size, threadsPerBlock>>>(d_mat.gpu,d_vettore,d_result_vettore,hack,matHll->numBlocks);


     hipError_t err = hipGetLastError();
        HANDLE_CUDA_ERROR(err);
        // calcolo il tempo del esecuznione
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        float time_ms;
        hipEventElapsedTime(&time_ms, start, stop);
        double time_sec = time_ms / 1000.0;
        *time=time_sec;


        hipError_t memcopy;
        memcopy=hipMemcpy(result->vettore, d_result_vettore, result->righe * sizeof(double), hipMemcpyDeviceToHost);
        HANDLE_CUDA_ERROR(memcopy);

        hipEventDestroy(start);
        hipEventDestroy(stop);
        hipFree(d_mat.pointersToGpu->values_flat);
        hipFree(d_mat.pointersToGpu->col_indices_flat );
        hipFree(d_result_vettore);
        hipFree(d_mat.pointersToGpu->block_offsets);
        hipFree(d_mat.pointersToGpu->block_nnz);
        hipFree(d_mat.pointersToGpu->block_rows);
        free(d_mat.pointersToGpu);

        *time=time_sec;


        
        return 0;

  
}