#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include "matriciOpp.h"
#include "cuda_alex.h"
#include <cstdint>


#include <hip/hip_runtime.h>
#include <stdio.h>
// Macro to check CUDA calls
#define CUDA_CHECK(call)                                                   \
    do {                                                                   \
        hipError_t err = (call);                                          \
        if (err != hipSuccess) {                                          \
            fprintf(stderr, "CUDA ERROR: %s (Error Code: %d) at %s:%d\n",  \
                    hipGetErrorString(err), err, __FILE__, __LINE__);     \
            exit(EXIT_FAILURE);                                            \
        }                                                                  \
    } while (0)

// Macro for safe hipMalloc
#define CUDA_MALLOC(ptr, size)                                  \
    do {                                                         \
        CUDA_CHECK(hipMalloc((void**)&ptr, (size)));            \
            fprintf(stderr,"Allocated %lu bytes at %p [%s:%d]\n",        \
                   (size_t)(size), (void*)(ptr), __FILE__, __LINE__); \
    } while (0)


// Macro for safe hipFree
#define CUDA_FREE(ptr)                                        \
    do {                                                     \
        if ((ptr) != NULL) {                                 \
            CUDA_CHECK(hipFree(ptr));                       \
            fprintf(stderr,"Freed memory at %p [%s:%d]\n",           \
                   (void*)(ptr), __FILE__, __LINE__);        \
            (ptr) = NULL;  /* Avoid dangling pointer */      \
        }                                                   \
    } while (0)

// Macro for safe hipMemcpy
#define CUDA_MEMCPY(dst, src, size, direction)                           \
    do {                                                                 \
        CUDA_CHECK(hipMemcpy((dst), (src), (size), (direction)));       \
        fprintf(stderr,"[CUDA MEMCPY] %lu bytes from %p to %p (Dir: %d) [%s:%d]\n", \
               (size_t)(size), (void*)(src), (void*)(dst), (direction), __FILE__, __LINE__); \
    } while (0)


void checkerror(){
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("Kernel launch failed: %s\n", hipGetErrorString(error));
    }
}
// Macro for safe hipMemcpy
#define CUDA_TIME(operation,time)                           \
    { \
    hipEvent_t start, stop;                               \
    CUDA_CHECK(hipEventCreate(&start));                \
    CUDA_CHECK(hipEventCreate(&stop));                 \
    CUDA_CHECK(hipEventRecord(start, 0));              \
    operation; \
    CUDA_CHECK(hipEventRecord(stop, 0)); \
    CUDA_CHECK(hipDeviceSynchronize());\
    float seconds = 0;  \
    CUDA_CHECK(hipEventElapsedTime(&seconds, start, stop)); \
    seconds=seconds/1000.0;\
    CUDA_CHECK(hipEventDestroy(start));\
    CUDA_CHECK(hipEventDestroy(stop));\
    (*time)=seconds;\
    } 



void allocateAndCopyMatriceCsrGpu( MatriceCsr *orgi,  MatriceCsr **mat, int coal) {
        hipMalloc((void**)mat, sizeof(MatriceCsr));
    
        unsigned int *d_iRP = NULL;
        unsigned int *d_jValori = NULL;
        double *d_valori = NULL;
    
        size_t size_iRP = (orgi->height + 1) * sizeof(unsigned int);
        if(coal==1){
            size_iRP=(orgi->height*2 ) * sizeof(unsigned int);
        }
        size_t size_jValori = orgi->nz * sizeof(unsigned int);
        size_t size_valori = orgi->nz * sizeof(double);
    
        CUDA_MALLOC(d_iRP, size_iRP);
        CUDA_MALLOC(d_jValori, size_jValori);
        CUDA_MALLOC(d_valori, size_valori);
    
        CUDA_MEMCPY(d_iRP, orgi->iRP, size_iRP, hipMemcpyHostToDevice);
        CUDA_MEMCPY(d_jValori, orgi->jValori, size_jValori, hipMemcpyHostToDevice);
        CUDA_MEMCPY(d_valori, orgi->valori, size_valori, hipMemcpyHostToDevice);
    
        MatriceCsr temp = *orgi;
        temp.iRP = d_iRP;
        temp.jValori = d_jValori;
        temp.valori = d_valori;
    
        CUDA_MEMCPY(*mat, &temp, sizeof(MatriceCsr), hipMemcpyHostToDevice);
}


void freeMatriceCsrGpu(MatriceCsr **mat_gpu) {
    if (mat_gpu == NULL || *mat_gpu == NULL) {
        return;
    }

    MatriceCsr temp_host;

    CUDA_CHECK(hipMemcpy(&temp_host, *mat_gpu, sizeof(MatriceCsr), hipMemcpyDeviceToHost));

    CUDA_FREE(temp_host.iRP);
    CUDA_FREE(temp_host.jValori);
    CUDA_FREE(temp_host.valori);

    MatriceCsr *ptr_to_free = *mat_gpu;
    CUDA_FREE(ptr_to_free);

    *mat_gpu = NULL;
}


void copyVectorBackToHost(Vector *cpu, Vector *gpu) {
    Vector temp;
    CUDA_MEMCPY(&temp, gpu, sizeof(Vector), hipMemcpyDeviceToHost);

    size_t size_vettore = cpu->righe * sizeof(double);
    CUDA_MEMCPY(cpu->vettore, temp.vettore, size_vettore, hipMemcpyDeviceToHost);
}
void allocateAndCopyVector(Vector *cpu, Vector **gpu) {
    CUDA_CHECK(hipMalloc((void**)gpu, sizeof(Vector)));

    double *d_vettore = NULL;
    size_t size_vettore = cpu->righe * sizeof(double);
    CUDA_MALLOC(d_vettore, size_vettore);
    CUDA_MEMCPY(d_vettore, cpu->vettore, size_vettore, hipMemcpyHostToDevice);

    Vector temp = *cpu;
    temp.vettore = d_vettore;

    CUDA_MEMCPY(*gpu, &temp, sizeof(Vector), hipMemcpyHostToDevice);
}
void freeVectorGpu(Vector **vec_gpu) {
    if (vec_gpu == NULL || *vec_gpu == NULL) {
        return;
    }

    Vector temp_host;

    CUDA_CHECK(hipMemcpy(&temp_host, *vec_gpu, sizeof(Vector), hipMemcpyDeviceToHost));

    CUDA_FREE(temp_host.vettore);

    Vector *ptr_to_free = *vec_gpu;
    CUDA_FREE(ptr_to_free);

    *vec_gpu = NULL;
}


void vectorMultiplySerial(Vector *a, Vector* b, Vector * result) {
    for (int i = 0; i < a->righe; ++i) {
        result->vettore[i] = a->vettore[i] * b->vettore[i];
    }
}

__global__ void csr_matvec_mul(MatriceCsr *d_mat, Vector *d_vec, Vector *d_result) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < d_mat->height) {
        double sum = 0.0;
        for (int j = d_mat->iRP[row]; j < d_mat->iRP[row + 1]; j++) {
            sum += d_mat->valori[j] * d_vec->vettore[d_mat->jValori[j]];
        }
        d_result->vettore[row] = sum;
    }
}


__inline__ __device__
double warpReduceSum(double val) {
    #pragma unroll
    for (int offset = 16; offset > 0; offset /= 2)
        val += __shfl_down_sync(0xFFFFFFFF, val, offset);
    return val;
}

__global__ void crs_mat_32_way(MatriceCsr *d_mat, Vector *d_vec, Vector *d_result) {
    int id = blockIdx.x * blockDim.x + threadIdx.x; // id
    int realRow = id >> 5; // Check row number dividing id % number of thread per warp 2^5
    int position = id & 31; // get position inside warp

    if (realRow >= d_mat->height) return; //exit if id is outisde of lines range
    int base = d_mat->iRP[realRow]; //start of array
    int rowDim = d_mat->iRP[realRow + 1] - base;
    double sum = 0.0;
    #pragma unroll
    for (int i = 0; (i + 1) * 32 <= rowDim; ++i) {
        int col_index = d_mat->jValori[base + i * 32 + position];
        double  matVal= d_mat->valori[base + i * 32 + position];
        double vectVal= d_vec->vettore[col_index];
        sum=fma(matVal,matVal,sum);
    }

    int remaining = rowDim % 32;
    if (remaining > 0) {
        int start_of_remaining = base + rowDim - remaining;
        if (position < remaining) {
            int col_index = d_mat->jValori[start_of_remaining + position];
            double  matVal= d_mat->valori[start_of_remaining + position];
            double vectVal= d_vec->vettore[col_index];
            sum=fma(matVal,matVal,sum);
        }
    }
    sum = warpReduceSum(sum);
    if (position == 0) {
        d_result->vettore[realRow] = sum; // Explicit cast if d_result is float
    }
}

__global__ void crs_mat_32_way_coal(MatriceCsr * __restrict__ d_mat, Vector * __restrict__ d_vec, Vector *d_result) {
    int id = blockIdx.x * blockDim.x + threadIdx.x; // id
    int realRow = id >> 5; // Check row number dividing id % number of thread per warp 2^5
     // get position inside warp

     //exit if id is outisde of lines range

    const int2* input_vec_ptr = reinterpret_cast<const int2*>(d_mat->iRP);
    int2 loaded_ints = input_vec_ptr[realRow];
    int base = loaded_ints.x; //start of array
    int rowDim = loaded_ints.y ;
    if (realRow >= d_mat->height) return;
    double sum = 0.0;
    int position = id & 31;
    rowDim-= base;
    for (int i = 0; (i + 1) * 32 <= rowDim; ++i) {
        int col_index = d_mat->jValori[base + i * 32 + position];
        double  matVal= d_mat->valori[base + i * 32 + position];
        double vectVal= d_vec->vettore[col_index];
        sum += matVal*vectVal;
    }

    int remaining = rowDim % 32;
    if (remaining > 0) {
        int start_of_remaining = base + rowDim - remaining;
        if (position < remaining) {
            int col_index = d_mat->jValori[start_of_remaining + position];
            double  matVal= d_mat->valori[start_of_remaining + position];
            double vectVal= d_vec->vettore[col_index];
            sum += matVal*vectVal;
        }
    }
    sum = warpReduceSum(sum);
    if (position == 0) {
        d_result->vettore[realRow] = sum; // Explicit cast if d_result is float
    }
}


__global__ void vectorMultiply(Vector *a, Vector *b, Vector *result) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < a->righe) {
        result->vettore[idx] = a->vettore[idx] * b->vettore[idx];
    }
}



int multCudaCSRKernelWarp(MatriceCsr *mat,Vector *vector,Vector *result,double *time,unsigned int threadsPerBlock){
    
    MatriceCsr *matG;
    allocateAndCopyMatriceCsrGpu(mat,&matG,0);
    Vector *vectorG;
    Vector *resultG;
    allocateAndCopyVector(vector,&vectorG);
    allocateAndCopyVector(result,&resultG);

    unsigned int rows=mat->height;
    int N = vector->righe*32;
    N=N+(threadsPerBlock-N%threadsPerBlock);
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
   
    CUDA_TIME((crs_mat_32_way<<<blocksPerGrid,threadsPerBlock>>>(matG, vectorG, resultG)),time);
    //CUDA_CHECK(hipDeviceSynchronize());
    
    copyVectorBackToHost(result,resultG);
    freeVectorGpu(&vectorG);
    freeVectorGpu(&resultG);
    freeMatriceCsrGpu(&matG);
}

int multCudaCSRKernelWarpCoal(MatriceCsr *mat,Vector *vector,Vector *result,double *time,unsigned int threadsPerBlock){
    
    MatriceCsr *matG;
    allocateAndCopyMatriceCsrGpu(mat,&matG,1);
    Vector *vectorG;
    Vector *resultG;
    allocateAndCopyVector(vector,&vectorG);
    allocateAndCopyVector(result,&resultG);

    unsigned int rows=mat->height;
    int N = vector->righe*32;
    N=N+(threadsPerBlock-N%threadsPerBlock);
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
   
    CUDA_TIME((crs_mat_32_way_coal<<<blocksPerGrid,threadsPerBlock>>>(matG, vectorG, resultG)),time);
    //CUDA_CHECK(hipDeviceSynchronize());
    
    copyVectorBackToHost(result,resultG);
    freeVectorGpu(&vectorG);
    freeVectorGpu(&resultG);
    freeMatriceCsrGpu(&matG);
}


int multCudaCSRKernelLinear(MatriceCsr *mat,Vector *vector,Vector *result,double *time,unsigned int threadsPerBlock){
    
    MatriceCsr *matG;
    allocateAndCopyMatriceCsrGpu(mat,&matG,0);
    Vector *vectorG;
    Vector *resultG;
    allocateAndCopyVector(vector,&vectorG);
    allocateAndCopyVector(result,&resultG);

    unsigned int rows=mat->height;
    int N = vector->righe;
    N=N+(threadsPerBlock-N%threadsPerBlock);
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
   
    CUDA_TIME((csr_matvec_mul<<<blocksPerGrid,threadsPerBlock>>>(matG, vectorG, resultG)),time);
    //CUDA_CHECK(hipDeviceSynchronize());
    
    copyVectorBackToHost(result,resultG);
    freeVectorGpu(&vectorG);
    freeVectorGpu(&resultG);
    freeMatriceCsrGpu(&matG);
}



void testVectors(int rows){
Vector *vector1;
Vector *vector2;
Vector *result;
Vector *resultSerial;
int seed = 42;
int num=1000000;
generate_random_vector(seed, num, &vector1) ;
generate_random_vector(seed, num, &vector2) ;
generateEmpty(num,&result);
generateEmpty(num,&resultSerial);

Vector *vector1G;
Vector *vector2G;
Vector *resultG;
allocateAndCopyVector(vector1,&vector1G);
allocateAndCopyVector(vector2,&vector2G);
allocateAndCopyVector(result,&resultG);

int N = vector1->righe;
int threadsPerBlock = 32;
int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
//printf("%d\n",blocksPerGrid);
hipEvent_t start, stop;
CUDA_CHECK(hipEventCreate(&start));
CUDA_CHECK(hipEventCreate(&stop));
CUDA_CHECK(hipEventRecord(start, 0));
vectorMultiply<<<blocksPerGrid, threadsPerBlock>>>(vector1G, vector2G, resultG);
CUDA_CHECK(hipEventRecord(stop, 0));
CUDA_CHECK(hipDeviceSynchronize());
float seconds = 0;//in milli secondi inizialmente
CUDA_CHECK(hipEventElapsedTime(&seconds, start, stop));
// Clean up events
seconds=seconds/1000.0;
CUDA_CHECK(hipEventDestroy(start));
CUDA_CHECK(hipEventDestroy(stop));
printf("giga flops %f\n",vector1->righe/(seconds*1000000000.0));
copyVectorBackToHost(result,resultG);
clock_t t;
t = clock();
vectorMultiplySerial(vector1,vector2,resultSerial);
t = clock() - t;
double execTime = ((double)t) / CLOCKS_PER_SEC; // in seconds
printf("giga flops %f\n",vector1->righe/(execTime*1000000000.0));
int areEq=areVectorsEqual(result,resultSerial);
printf("are equal(0 yes)? %d\n",areEq);
return ;

}

